#include "hip/hip_runtime.h"

#include <iostream>
#include <iomanip>
#include "kernels.cu"
#include "prep.cu"

#include <fstream>
using namespace std;

int main(int argc, char** argv) {
  // file pointer
  fstream fout;
  fout.open("data4.csv", ios::out | ios::app);
  std::ofstream myfile;
  myfile.open ("Messungen2.csv");

  // t
  int t = 1024;
  // i to calculate matrix size
  int i=16;




  //überprüfe, ob Eingabe gemacht wurde, falls ja verwende diese
  if (argc>1)
  {
    i=atoi(argv[1]);
  }
  int size = 1024 * i;

  //Kernel Konfiuration
  dim3 block_dim(32,32);
  dim3 grid_dim(size/block_dim.x, size/block_dim.y);


  // Datenfelder anlegen
  DTYPE *y_dev, *b_dev, *a_dev, *x_dev;
  DTYPE *yd_host, *yh_host, *a_host, *x_host;
  

  // allocate pointers for host
  x_host = (DTYPE *)malloc(size * sizeof(DTYPE));
  yh_host = (DTYPE *)malloc(size * sizeof(DTYPE));
  a_host = (DTYPE *)malloc(size * size * sizeof(DTYPE));
  yd_host = (DTYPE *)malloc(size * sizeof(DTYPE));

  // Matrix und Vektor füllen
  fillVector(x_host, size);
  fillMatrix(a_host, size);

  //cuda Memory
  hipMalloc((void **)&y_dev, size * sizeof(DTYPE));
  hipMalloc((void **)&b_dev, size * size * sizeof(DTYPE));
  hipMalloc((void **)&a_dev, size * size * sizeof(DTYPE));
  hipMalloc((void **)&x_dev, size * sizeof(DTYPE));
 

  hipEvent_t start,end;
  struct timespec start_h,end_h;

  // Zeiten auf numm setzen
  float hostToDevTime=0.0;
  float devToHostTime= 0.0;
  float hostTime = 0.0;
  float atomicKernelTime = 0.0;
  float sharedMemKernelTime = 0.0;
  float sharedMemAtomicAddKernelTime = 0.0;
  float kernelIntraGridTime =0.0;
  
  // events for time measurement
  hipEventCreate(&start);
  hipEventCreate(&end);

  //start time measurement
  hipEventRecord(start);

  // Daten von Host zu device kopieren
  hipMemcpy(a_dev, a_host, size * size * sizeof(DTYPE),hipMemcpyHostToDevice);
  hipMemcpy(x_dev, x_host, size * sizeof(DTYPE), hipMemcpyHostToDevice);

  //Zeitmessung beenden
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&hostToDevTime, start, end);

  printf("Zeit um Daten von Host zu Device zu kopieren (ms): %f\n", hostToDevTime);
  printf("In Sekunden: %f\n",hostToDevTime/1000);
  printf("\n");

  //check for errorors
  hipError_t error=hipGetLastError();
  if (error!=hipSuccess)
  {
    printf("Fehler im Speichermanagement!!!: %s (%i)\n",hipGetErrorString(error),error); //TODO
    return(-1);
  }
  


  //############################ CPU Zeitmessung ############################
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start_h);
  hostAx(a_host, x_host, yh_host, size);
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&end_h);
  hostTime=(double)((end_h.tv_nsec+end_h.tv_sec*1E9) - (start_h.tv_nsec+start_h.tv_sec*1E9))/1E6;
  printf("Zeit auf CPU in Sekunden: %f\n", hostTime/1000);
  printf("\n");
  //####################################################################################



  //######################## Shared Memory Kernel (Kernel 1) ###############################
  //set chache configuration
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelRed), hipFuncCachePreferEqual);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelRed), hipFuncCachePreferShared);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelRed), hipFuncCachePreferL1);

  hipMemset(y_dev,0,size*sizeof(DTYPE));
  int sdata_sz = (block_dim.x*block_dim.y)*sizeof(DTYPE);
  hipEventRecord(start);
  // memory kernels ln(size) mal aufrufen
  for(int i = 0; i < floor(log2((DTYPE)size)); i++){
    kernelRed<<<grid_dim, block_dim, sdata_sz>>>(a_dev, x_dev, y_dev, b_dev, size, i); 
  }
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&sharedMemKernelTime, start, end);
  hipDeviceSynchronize();

  error=hipGetLastError();
  if (error!=hipSuccess)
  {
    printf("Fehler im Shared Memory Kernel!!!: %s (%i)\n",hipGetErrorString(error),error);
    return(-1);
  }

  hipMemcpy(yd_host, y_dev, size * sizeof(DTYPE), hipMemcpyDeviceToHost);
  correctness(a_host, yd_host, yh_host, x_host, size);
  printf("Time Kernel 1 (seconds): %f\n", sharedMemKernelTime/1000);
  printf("\n");
  //########################################################################################


  //#################### Kernel 2.1 - shared memory with atomic operations at the end of a block_dim ########################
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelRedAtomicAdd), hipFuncCachePreferEqual);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelRedAtomicAdd), hipFuncCachePreferShared);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelRedAtomicAdd), hipFuncCachePreferL1);
  hipMemset(y_dev,0,size*sizeof(float));
  hipEventRecord(start);
  kernelRedAtomicAdd<<<grid_dim, block_dim,sdata_sz>>>(a_dev, x_dev, y_dev, size);
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&sharedMemAtomicAddKernelTime
, start, end);
  hipDeviceSynchronize();

  error=hipGetLastError();
  if (error!=hipSuccess)
  {
    printf("Error in Kernel 2.1 !!!: %s (%i)\n",hipGetErrorString(error),error);
    return(-1);
  }
  hipEventRecord(start);
  hipMemcpy(yd_host, y_dev, size * sizeof(DTYPE), hipMemcpyDeviceToHost);
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&devToHostTime, start, end);

  //Ergebnis Überprüfen
  correctness(a_host, yd_host, yh_host, x_host, size);
  //Zeit ausgeben
  printf("Zeit Kernel 2.1 (ms): %f\n", sharedMemAtomicAddKernelTime
);
  printf("in Sekunden:%f\n",sharedMemAtomicAddKernelTime
/1000);
  printf("\n");
  //########################################################################################

  // ################## Kernel 2.2 - all operations atomic ######################
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelAtomic), hipFuncCachePreferEqual);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelAtomic), hipFuncCachePreferShared);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelAtomic), hipFuncCachePreferL1);
  hipMemset(y_dev,0,size*sizeof(float));
  hipEventRecord(start);
  kernelAtomic<<<grid_dim, block_dim>>>(a_dev, x_dev, y_dev, size);
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&atomicKernelTime, start, end);
  hipDeviceSynchronize();

  error=hipGetLastError();
  if (error!=hipSuccess)
  {
    printf("Fehler in Kernel 2.2 !!! : %s (%i)\n",hipGetErrorString(error),error);
    return(-1);
  }

  hipMemcpy(yd_host, y_dev, size * sizeof(DTYPE), hipMemcpyDeviceToHost);

  //Ergebnis Überprüfen
  correctness(a_host, yd_host, yh_host, x_host, size);

  //Zeit ausgeben
  printf("Zeit Kernel 2.2 (ms): %f\n", atomicKernelTime);
  printf("in Sekunden: %f\n", atomicKernelTime/1000);
  printf("\n");
  //####################################################################################-


  //################## Third Kernel: Intra Grid Groups (Kernel 3) #####################
//hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelIntraGrid), hipFuncCachePreferEqual);
//hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelIntraGrid), hipFuncCachePreferShared);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelIntraGrid), hipFuncCachePreferL1);
  


  block_dim.x = t;
  block_dim.y = 1;

  DTYPE* buf;
  hipMalloc((void **)&buf, size * sizeof(DTYPE));
  hipMemset(y_dev,0,size*sizeof(float));
  hipMemset(buf,0,size*sizeof(float));

  //Berechnen der device occupancy, um zu wissen, wie viele Blöcke gleichzeitig ausgeführt werden können
  int numBlocksPerSm;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, kernelIntraGrid, t, t*sizeof(DTYPE));

  int num_sums = deviceProp.multiProcessorCount;
  int n_blocks = min(numBlocksPerSm*num_sums, (size+t-1)/t);

  grid_dim.x = n_blocks;
  grid_dim.y = 1;
 
  void *params[] = {(void *)&a_dev, (void *)&x_dev, (void *)&y_dev,(void*)&buf, (void *)&size};
  sdata_sz = t*sizeof(DTYPE);
  hipEventRecord(start);
  
  hipLaunchCooperativeKernel((void *)kernelIntraGrid, grid_dim, block_dim, params, sdata_sz, NULL);
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&kernelIntraGridTime, start, end);
 
  hipMemcpy(yd_host, y_dev, size * sizeof(DTYPE), hipMemcpyDeviceToHost);

  
  error=hipGetLastError();
  if (error!=hipSuccess)
  {
    printf("Fehler im intra Grid Kernel!!! : %s (%i)\n",hipGetErrorString(error),error);
    return(-1);
  }

  hipMemcpy(yd_host, y_dev, size * sizeof(DTYPE), hipMemcpyDeviceToHost);

  //Ergebnis Überprüfen
  correctness(a_host, yd_host, yh_host, x_host, size);

  //Zeit ausgeben
  printf("Zeit Intra Grid Kernel (ms): %f\n", kernelIntraGridTime);
  printf("in Sekunden: %f\n", kernelIntraGridTime/1000);
  printf("\n");
  //#############################################################################################################

  // Zeiten in Datei schreiben
  string cache_config ="PrefereNone";
  string gpu = "GPU01";
  //fout << "ArraySize; tX; tY; GridSizeX; GridSizeY; Kernel_1; Kernel_2.1; Kernel_2.2; Kernel_1.2; HostTime; HostToDevice; DeviceToHost; Chache_Configuration;GPU \n";
  //fout <<  size << ";" << block_dim.x << ";" <<block_dim.y<< ";"<<grid_dim.x<< ";"<< grid_dim.y<< ";"<< sharedMemKernelTime << ";"<<sharedMemAtomicAddKernelTime
 //<< ";"<< atomicKernelTime << ";"<< kernelIntraGridTime<< ";"<< hostTime<< ";"<< hostToDevTime<< ";"<< devToHostTime <<";" << cache_config << ";"<<gpu <<"\n";

  
  myfile.close();

  //destroy cuda events
  hipEventDestroy(start);
  hipEventDestroy(end);

  //free memory for device and host
  hipFree(a_dev);
  hipFree(y_dev);
  hipFree(x_dev);
  hipFree(b_dev);

  free(a_host);
  free(yd_host);
  free(yh_host);
  free(x_host);
  return 0;
}
